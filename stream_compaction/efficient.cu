#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include <hip/device_functions.h>

#define NUM_BANKS 32
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n)(((n) >> NUM_BANKS) + ((n) >> (2 * LOG_NUM_BANKS)))

#define MAX_THREADS_PER_BLOCK 1024
#define MAX_ELEMENTS_PER_BLOCK (MAX_THREADS_PER_BLOCK * 2)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernPrescan(int n, int* odata, int* idata, int* sum) {
            extern __shared__ int temp[MAX_THREADS_PER_BLOCK];
            int idx = threadIdx.x;
            int bid = blockIdx.x;
            int blockOffset = bid * MAX_ELEMENTS_PER_BLOCK;
            int leafNum = MAX_ELEMENTS_PER_BLOCK;

            int offset = 1;
            int ai = idx;
            int bi = idx + (leafNum >> 1);
            int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
            int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
            temp[ai + bankOffsetA] = ai + bankOffsetA < n ? idata[ai] : 0;
            temp[bi + bankOffsetB] = bi + bankOffsetB < n ? idata[bi] : 0;
            
            for (int d = leafNum >> 1; d > 0; d >>= 1) {
                __syncthreads();
                if (idx < d) {
                    int ai = offset * (2 * idx + 1) - 1;
                    int bi = offset * (2 * idx + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    temp[bi] += temp[ai];
                }
                offset *= 2;
            }

            __syncthreads();
            if (idx == 0) {
                // The last element in a block
                int index = leafNum - 1 + CONFLICT_FREE_OFFSET(n - 1);
                sum[bid] = temp[index];
                temp[index] = 0;
            }

            for (int d = 1; d < leafNum; d *= 2) {
                offset >>= 1;
                __syncthreads();
                if (idx < d) {
                    int ai = offset * (2 * idx + 1) - 1;
                    int bi = offset * (2 * idx + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();
            if (ai + bankOffsetA < n)
                odata[ai] = temp[ai + bankOffsetA];

            if (bi + bankOffsetB < n)
                odata[bi] = temp[bi + bankOffsetB];
        }

        __global__ void kernAdd(int n, int* valus, int* prefix_sum)
        {
            int idx = threadIdx.x;
            int bid = blockIdx.x;
            int block_offset = bid * MAX_ELEMENTS_PER_BLOCK;
            int ai = idx + block_offset;
            int bi = idx + (MAX_ELEMENTS_PER_BLOCK >> 1) + block_offset;

            if (ai < n)
            {
                valus[ai] += prefix_sum[bid];
            }
            if (bi < n)
            {
                valus[bi] += prefix_sum[bid];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_odata;
            int* dev_idata;
            int* dev_sum;
            int blockNum = (n + MAX_ELEMENTS_PER_BLOCK - 1) / MAX_ELEMENTS_PER_BLOCK;
            int sizeInBytes = n * sizeof(int);
            hipMalloc((void**)&dev_odata, sizeInBytes);
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_idata, sizeInBytes);
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_sum, blockNum * sizeof(int));
            checkCUDAError("hipMalloc dev_sum failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to dev_idata failed!");
            
            dim3 fullBlocksPerGrid(blockNum);

            timer().startGpuTimer();
            kernPrescan << <fullBlocksPerGrid, MAX_THREADS_PER_BLOCK>> > (n, dev_odata, dev_idata, dev_sum);
            checkCUDAError("kernPrescan executed failed!");

            hipDeviceSynchronize();
            if (blockNum != 1) {
                kernAdd << <fullBlocksPerGrid, MAX_THREADS_PER_BLOCK >> > (n, dev_odata, dev_sum);
                checkCUDAError("kernAdd executed failed!");
            }

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy to odata failed!");

            timer().endGpuTimer();

            hipFree(dev_odata);
            hipFree(dev_idata);
            checkCUDAError("hipFree failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
